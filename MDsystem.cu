#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "math_lib.cuh"
#include "math_func.cuh"

#include "MDsystem.cuh"

#ifndef H5_NO_NAMESPACE
    using namespace H5;
#endif

//******************* MDcell methods ***************************//

__constant__ Cell dc_Cell;

MDcell* MDcell::m_pInstance = NULL;

MDcell* MDcell::Instance()
{
	static Cleanup cleanup;
	if (!m_pInstance)
		m_pInstance = new MDcell();
	return m_pInstance;
}

MDcell::Cleanup::~Cleanup()
{
	delete MDcell::m_pInstance;
	MDcell::m_pInstance = NULL;	
}

MDcell::MDcell() : MDinterface("Cell")
{
	List.push_back(MDdata(&h_Data.H, 1, "Cell tensor", HDF5interface::H5matrix9()));

	for (int igpu =  dev_ptr.size(); igpu--;)	
	{
		checkCudaErrors(hipSetDevice(igpu)); 
		checkCudaErrors(hipGetSymbolAddress((void**)&dev_ptr[igpu], dc_Cell));
	}
}

void MDcell::Set(double3 Box, int ngpu)
{
	m_ngpu = ngpu;
	m_set = true;
	Set(Box);
}

void MDcell::Set(matrix9 H, int ngpu)
{
	m_ngpu = ngpu;
	m_set = true;
	Set(H);
}

void MDcell::Set(double3 Box) 
{
	h_Data.V = Box.x*Box.y*Box.z;
	h_Data.rV = 1.0/h_Data.V;
	h_Data.Box = Box;
	h_Data.rBox = inverse(h_Data.Box);
	h_Data.H = make_diagonal(Box);
	h_Data.rH = inverse(h_Data.H);
	UpdateDevice();
}

void MDcell::Set(matrix9 H) 
{
	matrix9 Ht = transpose(H);
	h_Data.V = det(H);
	h_Data.rV = 1.0/h_Data.V;
	h_Data.H = H;
	h_Data.rH = inverse(H);
	h_Data.Box = make_double3(norm(Ht.X), norm(Ht.Y), norm(Ht.Z));
	h_Data.rBox = inverse(h_Data.Box);	
	UpdateDevice();
}

void MDcell::ScaleH(matrix9 S) {
	matrix9 H = MultiplyByElement(h_Data.H, S);
	Set(H);	
}

void MDcell::UpdateDevice()
{
	for (int igpu = m_ngpu; igpu--;)	
	{
		checkCudaErrors(hipSetDevice(igpu)); 
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dc_Cell), &h_Data, sizeof(Cell)));
	}
}

double3 MDcell::CellWidth() {
	double3 W;
	matrix9 Ht = transpose(h_Data.H);
	W.x = h_Data.V / norm(cross(Ht.Y, Ht.Z));
	W.y = h_Data.V / norm(cross(Ht.Z, Ht.X));
	W.z = h_Data.V / norm(cross(Ht.X, Ht.Y));
	return W;
}

double3 MDcell::CellAngles() {
	double3 W;
	matrix9 Ht = transpose(h_Data.H);
	W.x = angle(Ht.X, Ht.Y);
	W.y = angle(Ht.Y, Ht.Z);
	W.z = angle(Ht.X, Ht.Z);
	return W;
}

double MDcell::GetMinLength() {	
	return min(CellWidth()); 
}

double MDcell::GetMaxLength() {
	return max(CellWidth()); 
}

double3 MDcell::GetLengthRatio() {
	double l = GetMinLength();
	return h_Data.Box/l;
}

void MDcell::Print() {printf("Box size: %f %f %f\n\n", h_Data.Box.x, h_Data.Box.y, h_Data.Box.z);}
